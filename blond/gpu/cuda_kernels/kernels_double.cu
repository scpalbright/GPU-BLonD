#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <hiprand/hiprand_kernel.h>
#define REDUCE(a, b) (a+b)
#define BLOCK_SIZE 512
#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286
#define PI_DIV_2 3.141592653589793238462643383279502884197169399375105820974944592307816406286/2

// Note that any atomic operation can be implemented based on atomicCAS() (Compare And Swap). 
// For example, atomicAdd() for double-precision floating-point numbers is not 
// available on devices with compute capability lower than 6.0 but it can be implemented 
// as follows: 
#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


extern "C"
__global__ void gpu_losses_longitudinal_cut(
    double *dt,
    double *dev_id,
    const int size,
    const double min_dt,
    const double max_dt)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if ((dt[i] - min_dt) * (max_dt - dt[i]) < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void gpu_losses_energy_cut(
    double *dE,
    double *dev_id,
    const int size,
    const double min_dE,
    const double max_dE)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if ((dE[i] - min_dE) * (max_dE - dE[i]) < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void gpu_losses_below_energy(
    double *dE,
    double *dev_id,
    const int size,
    const double min_dE)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if (dE[i] - min_dE < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void cuinterp(double *x,
                         int x_size,
                         double *xp,
                         int xp_size,
                         double *yp,
                         double *y,
                         double left,
                         double right)
{
    if (left == 0.12345)
        left = yp[0];
    if (right == 0.12345)
        right = yp[xp_size - 1];
    double curr;
    int lo;
    int mid;
    int hi;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < x_size; i += blockDim.x * gridDim.x) {
        //need to find the right bin with binary search
        // looks like bisect_left
        curr = x[i];
        hi = xp_size;
        lo = 0;
        while (lo < hi) {
            mid = (lo + hi) / 2;
            if (xp[mid] < curr)
                lo = mid + 1;
            else
                hi = mid;
        }
        if (lo == xp_size)
            y[i] = right;
        else if (xp[lo - 1] == curr)
            y[i] = yp[i];
        else if (lo <= 1)
            y[i] = left;
        else {
            y[i] = yp[lo - 1] +
                   (yp[lo] - yp[lo - 1]) * (x[i] - xp[lo - 1]) /
                   (xp[lo] - xp[lo - 1]);
        }

    }
}

extern "C"
__global__ void cugradient(
    double x,
    int *y,
    double *g,
    int size)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid + 1; i < size - 1; i += blockDim.x * gridDim.x) {

        g[i] = (y[i + 1] - y[i - 1]) / (2 * x);
        // g[i] = (hs*hs*fd + (hd*hd-hs*hs)*fx - hd*hd*fs)/
        //     (hs*hd*(hd+hs));
    }
    if (tid == 0)
        g[0] = (y[1] - y[0]) / x;
    if (tid == 32)
        g[size - 1] = (y[size - 1] - y[size - 2]) / x;
}


extern "C"
__global__ void gpu_beam_fb_track_other(double *omega_rf,
                                        double *harmonic,
                                        double *dphi_rf,
                                        double *omega_rf_d,
                                        double *phi_rf,
                                        // double pi,
                                        double domega_rf,
                                        int size,
                                        int counter,
                                        int n_rf)
{
    double a, b, c;
    for (int i = threadIdx.x; i < n_rf; i += blockDim.x) {
        a = domega_rf * harmonic[i * size + counter] / harmonic[counter];
        b =  2.0 * PI * harmonic[size * i + counter] * (a + omega_rf[i * size + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        c = dphi_rf[i] +  b;
        omega_rf[i * size + counter] += a;
        dphi_rf[i] +=  b;
        phi_rf[size * i + counter] += c;
    }
}

extern "C"
__global__ void gpu_rf_voltage_calc_mem_ops(double *new_voltages,
        double *new_omega_rf,
        double *new_phi_rf,
        double *voltages,
        double *omega_rf,
        double *phi_rf,
        int start,
        int end,
        int step)
{
    int idx = 0;
    for (int i = threadIdx.x * step + start; i < end; i += blockDim.x * step) {
        new_voltages[idx] = voltages[i];
        new_omega_rf[idx] = omega_rf[i];
        new_phi_rf[idx] = phi_rf[i];
        idx++;
    }
} 

extern "C"
__global__ void halve_edges(double *my_array, int size) {
    //__shared__ my_sum;
    int tid = threadIdx.x;
    if (tid == 0) {
        my_array[0] = my_array[0] / 2.;
    }
    if (tid == 32) {
        my_array[size - 1] = my_array[size - 1] / 2.;
    }
}

extern "C"
__global__ void simple_kick(
    const double  *beam_dt,
    double        *beam_dE,
    const int n_rf,
    const double  *voltage,
    const double  *omega_RF,
    const double  *phi_RF,
    const int n_macroparticles,
    const double acc_kick
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double my_beam_dt;
    double sin_res;
    double dummy;
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        my_beam_dt = beam_dt[i];
        for (int j = 0; j < n_rf; j++) {
            sincos(omega_RF[j]*my_beam_dt + phi_RF[j], &sin_res, &dummy);
            beam_dE[i] += voltage[j] * sin_res;
        }
        beam_dE[i] += acc_kick;
    }
}

extern "C"
__global__ void rf_volt_comp(double *voltage,
                             double *omega_rf,
                             double *phi_rf,
                             double *bin_centers,
                             int n_rf,
                             int n_bins,
                             int f_rf,
                             double *rf_voltage)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__ double s[];
    if (threadIdx.x == 0){
        for (int j = 0; j < n_rf; j++) {
            s[j] = voltage[j];
            s[j + n_rf] = omega_rf[j];
            s[j + 2 * n_rf] = phi_rf[j];
        }
    }

    __syncthreads();
    for (int i = tid; i < n_bins; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_rf; j++)
            // rf_voltage[i] = voltage[j] * sin(omega_rf[j] * bin_centers[i] + phi_rf[j]);
            rf_voltage[i] = s[j] * sin(s[j+n_rf] * bin_centers[i] + s[j+2*n_rf]);
    }
}

extern "C"
__global__ void drift(double *beam_dt,
        const double  *beam_dE,
        const int solver,
        const double T0, const double length_ratio,
        const double alpha_order, const double eta_zero,
        const double eta_one, const double eta_two,
        const double alpha_zero, const double alpha_one,
        const double alpha_two,
        const double beta, const double energy,
        const int n_macroparticles)
{
    double T = T0 * length_ratio;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if ( solver == 0 )
    {
        double coeff = eta_zero / (beta * beta * energy);
        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
            beam_dt[i] += T * coeff * beam_dE[i];
    }

    else if ( solver == 1 )
    {
        const double coeff = 1. / (beta * beta * energy);
        const double eta0 = eta_zero * coeff;
        const double eta1 = eta_one * coeff * coeff;
        const double eta2 = eta_two * coeff * coeff * coeff;

        if (alpha_order == 0)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) - 1.);
        else if (alpha_order == 1)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]) - 1.);
        else
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]
                                         - eta2 * beam_dE[i] * beam_dE[i] * beam_dE[i]) - 1.);
    }

    else
    {

        const double invbetasq = 1 / (beta * beta);
        const double invenesq = 1 / (energy * energy);
        // double beam_delta;

        
        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)

        {

            double beam_delta = sqrt(1. + invbetasq *
                              (beam_dE[i] * beam_dE[i] * invenesq + 2.*beam_dE[i] / energy)) - 1.;

            beam_dt[i] += T * (
                              (1. + alpha_zero * beam_delta +
                               alpha_one * (beam_delta * beam_delta) +
                               alpha_two * (beam_delta * beam_delta * beam_delta)) *
                              (1. + beam_dE[i] / energy) / (1. + beam_delta) - 1.);

        }

    }    
    
}   



extern "C"
__global__ void histogram(double * input,
                          int * output, const double cut_left,
                          const double cut_right, const int n_slices,
                          const int n_macroparticles)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    double const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i = i + blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        atomicAdd(&(output[target_bin]), 1);
    }
}

extern "C"
__global__ void hybrid_histogram(double * input,
                                 int * output, const double cut_left,
                                 const double cut_right, const unsigned int n_slices,
                                 const int n_macroparticles, const int capacity)
{
    extern __shared__ int block_hist[];
    //reset shared memory
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    double const inv_bin_width = n_slices / (cut_right - cut_left);

    const int low_tbin = (n_slices / 2) - (capacity / 2);
    const int high_tbin = low_tbin + capacity;


    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        if (target_bin >= low_tbin && target_bin < high_tbin)
            atomicAdd(&(block_hist[target_bin - low_tbin]), 1);
        else
            atomicAdd(&(output[target_bin]), 1);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        atomicAdd(&output[low_tbin + i], block_hist[i]);
}


extern "C"
__global__ void sm_histogram(double * input,
                             int * output, const double cut_left,
                             const double cut_right, const unsigned int n_slices,
                             const int n_macroparticles)
{
    extern __shared__ int block_hist[];
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    double const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        atomicAdd(&(block_hist[target_bin]), 1);
    }
    __syncthreads();
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        atomicAdd(&output[i], block_hist[i]);
}


extern "C"
__global__ void lik_only_gm_copy(
    double *beam_dt,
    double *beam_dE,
    const double *voltage_array,
    const double *bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double *glob_voltageKick,
    double *glob_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);


    for (int i = tid; i < n_slices - 1; i += gridDim.x * blockDim.x) {
        glob_voltageKick[i] = charge * (voltage_array[i + 1] - voltage_array[i])
                              * inv_bin_width;
        glob_factor[i] = (charge * voltage_array[i] - bin_centers[i] * glob_voltageKick[i])
                         + acc_kick;
    }
}


extern "C"
__global__ void lik_only_gm_comp(
    double *beam_dt,
    double *beam_dE,
    const double *voltage_array,
    const double *bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double *glob_voltageKick,
    double *glob_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    int fbin;
    const double bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = floor((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1) && (fbin >= 0))
            beam_dE[i] += beam_dt[i] * glob_voltageKick[fbin] + glob_factor[fbin];
    }
}


extern "C"
__global__ void lik_drift_only_gm_comp(
    double *beam_dt,
    double *beam_dE,
    const double *voltage_array,
    const double *bin_centers,
    const double charge,
    const int n_slices,
    const int n_macroparticles,
    const double acc_kick,
    double *glob_voltageKick,
    double *glob_factor,
    const double T0, const double length_ratio,
    const double eta0, const double beta, const double energy
)
{
    const double T = T0 * length_ratio * eta0 / (beta * beta * energy);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    unsigned fbin;
    const double bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = (unsigned) floor((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1))
            beam_dE[i] += beam_dt[i] * glob_voltageKick[fbin] + glob_factor[fbin];
        // beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) -1.);
        beam_dt[i] += T * beam_dE[i];
    }
}

extern "C"
__global__ void beam_phase_v2(
    const double *bin_centers,
    const int *profile,
    const double alpha,
    const double *omega_rf_ar,
    const double *phi_rf_ar,
    const int ind,
    const double bin_size,
    double *array1,
    double *array2,
    const int n_bins)
{
    double omega_rf = omega_rf_ar[ind];
    double phi_rf = phi_rf_ar[ind];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double a, b;
    double sin_res, cos_res;
    for (int i = tid; i < n_bins; i += gridDim.x * blockDim.x) {
        a = omega_rf * bin_centers[i] + phi_rf;
        sincos(a, &sin_res, &cos_res);
        b = exp(alpha * bin_centers[i]) * profile[i];
        array1[i] = b * sin_res;
        array2[i] = b * cos_res;
    }
} 

extern "C" 
__global__ void beam_phase_sum(
    const double *ar1,
    const double *ar2,
    double *scoeff,
    double *coeff,
    int n_bins)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid == 0) {
        scoeff[0] = 0;
        coeff[0] = 0;
    }
    double my_sum_1 = 0;
    double my_sum_2 = 0;
    if (tid == 0) {
        my_sum_1 += ar1[0] / 2 + ar1[n_bins - 1] / 2;
        my_sum_2 += ar2[0] / 2 + ar2[n_bins - 1] / 2;
    }
    for (int i = tid + 1; i < n_bins - 1; i += gridDim.x * blockDim.x) {
        my_sum_1 += ar1[i];
        my_sum_2 += ar2[i];
    }
    atomicAdd(&(scoeff[0]), my_sum_1);
    atomicAdd(&(coeff[0]), my_sum_2);
    __syncthreads();
    if (tid == 0)
        scoeff[0] = scoeff[0] / coeff[0];

} 

extern "C"
__global__ void gpu_trapz_custom(
    double *y,
    double x,
    int sz,
    double *res)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double my_sum = 0;
    for (int i = tid; i < sz - 1; i += gridDim.x * blockDim.x)
        my_sum += (y[i] + y[i + 1]) * x / 2.0;

    atomicAdd(&(res[0]), my_sum);
}


extern "C" 
__global__ void gpu_trapz_stage1(double *out, double *y, double x, int sz,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + ((i < sz - 1) ? x * (y[i] + y[i + 1]) / 2.0 : 0.0);
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}



extern "C" 
__global__ void gpu_trapz_stage2(double *out, const double *pycuda_reduction_inp, double *y, double x, int sz,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + (pycuda_reduction_inp[i]);
        i = 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}


extern "C"
__global__ void mean_non_zeros_stage1(double *out, double *x, double *id,
                           unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + ((id[i] != 0) * x[i]);
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void mean_non_zeros_stage2(double *out, const double *pycuda_reduction_inp, double *x, double *id,
                           unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (512 >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (512 >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (512 >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (512 >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (512 >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (512 >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void stdKernel_stage1(double *out, double *x, double *y, double m,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, ((y[i] != 0) * (x[i] - m) * (x[i] - m)));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void stdKernel_stage2(double *out, const double *pycuda_reduction_inp, double *x, double *y, double m,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void sum_non_zeros_stage1(double *out, double *x,
                          unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, ((x[i] != 0)));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void sum_non_zeros_stage2(double *out, const double *pycuda_reduction_inp, double *x,
                          unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    double acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile double *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}



extern "C"
__global__ void gpu_copy_i2d(double *x, int *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = (double) y[i] * 1.0;
    }
    ;
}



extern "C"
__global__ void gpu_copy_d2d(double *x, double *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i];
    }
    ;
}



extern "C"
__global__ void gpu_complex_copy(pycuda::complex<double> *x, pycuda::complex<double> *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i];
    }
    ;
}



extern "C"
__global__ void gpu_diff(int *a, double *b, double c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] = (a[i + 1] - a[i]) / c;
    }
    ;
}



extern "C"
__global__ void set_zero_double(double *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}

extern "C"
__global__ void set_zero_float(float *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}


extern "C"
__global__ void set_zero_int(int *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}



extern "C"
__global__ void set_zero_complex64(pycuda::complex<float> *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}


extern "C"
__global__ void set_zero_complex128(pycuda::complex<double> *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}


extern "C"
__global__ void increase_by_value(double *x, double a, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] += a;
    }
    ;
}



extern "C"
__global__ void add_array(double *x, double *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] += y[i];
    }
    ;
}



extern "C"
__global__ void complex_mul(pycuda::complex<double> *x, pycuda::complex<double> *y, pycuda::complex<double> *z, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        z[i] = x[i] * y[i];
    }
    ;
}



extern "C"
__global__ void gpu_mul(double *x, double *y, double a, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = a * y[i];
    }
    ;
}



extern "C"
__global__ void gpu_copy_one(double *x, double *y, int ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[ind];
    }
    ;
}



extern "C"
__global__ void first_kernel_x(double *omega_rf, double *harmonic,  double domega_rf, int size, int counter, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
    }
    ;
}



extern "C"
__global__ void second_kernel_x(double *dphi_rf, double *harmonic, double *omega_rf, double *omega_rf_d, int size, int counter, double pi, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
    }
    ;
}



extern "C"
__global__ void third_kernel_x(double *x, double *y, int size_0, int counter, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i * size_0 + counter] += y[i];
    }
    ;
}



extern "C"
__global__ void indexing_double(double *out, double *in, int *ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        out[i] = in[ind[i]];
    }
    ;
}

// extern "C"
// __global__ void indexing_float(float *out, float *in, int *ind, long n)
// {
//     unsigned tid = threadIdx.x;
//     unsigned total_threads = gridDim.x * blockDim.x;
//     unsigned cta_start = blockDim.x * blockIdx.x;
//     unsigned i;
//     ;
//     for (i = cta_start + tid; i < n; i += total_threads)
//     {
//         out[i] = in[ind[i]];
//     }
//     ;
// }


extern "C"
__global__ void indexing_int(double *out, int *in, int *ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        out[i] = in[ind[i]];
    }
    ;
}



extern "C"
__global__ void sincos_mul_add(double *ar, double a, double b, double *s, double *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        sincos(a * ar[i] + b, &s[i], &c[i]);
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_2(double *ar, double a, double b, double *s, double *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        s[i] = cos(a * ar[i] + b - PI_DIV_2); c[i] = cos(a * ar[i] + b);
    }
    ;
}



extern "C"
__global__ void mul_d(double *a1, double *a2, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a1[i] *= a2[i];
    }
    ;
}



extern "C"
__global__ void add_kernel(double *a, double *b, double *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = b[i] + c[i];
    }
    ;
}



extern "C"
__global__ void first_kernel_tracker(double *phi_rf, double x, double *phi_noise, int len, int turn, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
    }
    ;
}



extern "C"
__global__ void second_kernel_tracker(double *phi_rf, double *omega_rf, double *phi_mod0, double *phi_mod1, int size, int turn, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
    }
    ;
}



extern "C"
__global__ void copy_column(double *x, double *y, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i * size + column];
    }
    ;
}



extern "C"
__global__ void rf_voltage_calculation_kernel(double *x, double *y, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i * size + column];
    }
    ;
}



extern "C"
__global__ void cavityFB_case(double *rf_voltage, double *voltage, double *omega_rf, double *phi_rf, double *bin_centers, double V_corr, double phi_corr, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        rf_voltage[i] = voltage[0] * V_corr * sin(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
    }
    ;
}



extern "C"
__global__ void bm_phase_exp_times_scalar(double *a, double *b, double c, int *d, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = exp(c * b[i]) * d[i];
    }
    ;
}



extern "C"
__global__ void bm_phase_mul_add(double *a, double b, double *c, double d, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = b * c[i] + d;
    }
    ;
}



extern "C"
__global__ void bm_sin_cos(double *a, double *b, double *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        sincos(a[i], &b[i], &c[i]);
    }
    ;
}



extern "C"
__global__ void d_multiply(double *a, double *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] *= b[i];
    }
    ;
}



extern "C"
__global__ void d_multscalar(double *a, double *b, double c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = c * b[i];
    }
    ;
}


extern "C"
__global__ void d_mul_int_by_scalar(int *a, double c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] *= c;
    }
    ;
}



extern "C"
__global__ void scale_kernel_int(int a, int *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}



extern "C"
__global__ void scale_kernel_double(double a, double *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}

extern "C"
__global__ void scale_kernel_float(float a, float *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}


extern "C"
__global__ void gpu_copy_i2d_range(double *x, int *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = (double) y[i] * 1.0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = (double) y[i] * 1.0;
        }
    }
    ;
}



extern "C"
__global__ void gpu_copy_d2d_range(double *x, double *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_complex_copy_range(pycuda::complex<double> *x, pycuda::complex<double> *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_diff_range(int *a, double *b, double c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] = (a[i + 1] - a[i]) / c;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] = (a[i + 1] - a[i]) / c;
        }
    }
    ;
}


extern "C"
__global__ void set_zero_float_range(float *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}

extern "C"
__global__ void set_zero_double_range(double *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}



extern "C"
__global__ void set_zero_int_range(int *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}


extern "C"
__global__ void set_zero_complex64_range(pycuda::complex<float> *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}


extern "C"
__global__ void set_zero_complex128_range(pycuda::complex<double> *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}



extern "C"
__global__ void increase_by_value_range(double *x, double a , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] += a;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] += a;
        }
    }
    ;
}



extern "C"
__global__ void add_array_range(double *x, double *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] += y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] += y[i];
        }
    }
    ;
}



extern "C"
__global__ void complex_mul_range(pycuda::complex<double> *x, pycuda::complex<double> *y, pycuda::complex<double> *z , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            z[i] = x[i] * y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            z[i] = x[i] * y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_mul_range(double *x, double *y, double a , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = a * y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = a * y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_copy_one_range(double *x, double *y, int ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[ind];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[ind];
        }
    }
    ;
}



extern "C"
__global__ void first_kernel_x_range(double *omega_rf, double *harmonic,  double domega_rf, int size, int counter , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
        }
    }
    ;
}



extern "C"
__global__ void second_kernel_x_range(double *dphi_rf, double *harmonic, double *omega_rf, double *omega_rf_d, int size, int counter, double pi , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        }
    }
    ;
}



extern "C"
__global__ void third_kernel_x_range(double *x, double *y, int size_0, int counter , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i * size_0 + counter] += y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i * size_0 + counter] += y[i];
        }
    }
    ;
}



extern "C"
__global__ void indexing_double_range(double *out, double *in, int *ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    ;
}



extern "C"
__global__ void indexing_int_range(double *out, int *in, int *ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_range(double *ar, double a, double b, double *s, double *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            sincos(a * ar[i] + b, &s[i], &c[i]);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            sincos(a * ar[i] + b, &s[i], &c[i]);
        }
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_2_range(double *ar, double a, double b, double *s, double *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            s[i] = cos(a * ar[i] + b - PI_DIV_2); c[i] = cos(a * ar[i] + b);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            s[i] = cos(a * ar[i] + b - PI_DIV_2); c[i] = cos(a * ar[i] + b);
        }
    }
    ;
}



extern "C"
__global__ void mul_d_range(double *a1, double *a2 , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a1[i] *= a2[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a1[i] *= a2[i];
        }
    }
    ;
}



extern "C"
__global__ void add_kernel_range(double *a, double *b, double *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = b[i] + c[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = b[i] + c[i];
        }
    }
    ;
}



extern "C"
__global__ void first_kernel_tracker_range(double *phi_rf, double x, double *phi_noise, int len, int turn , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
        }
    }
    ;
}



extern "C"
__global__ void second_kernel_tracker_range(double *phi_rf, double *omega_rf, double *phi_mod0, double *phi_mod1, int size, int turn , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
        }
    }
    ;
}



extern "C"
__global__ void copy_column_range(double *x, double *y, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    ;
}



extern "C"
__global__ void rf_voltage_calculation_kernel_range(double *x, double *y, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    ;
}



extern "C"
__global__ void cavityFB_case_range(double *rf_voltage, double *voltage, double *omega_rf, double *phi_rf, double *bin_centers, double V_corr, double phi_corr, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            rf_voltage[i] = voltage[0] * V_corr * sin(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            rf_voltage[i] = voltage[0] * V_corr * sin(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
        }
    }
    ;
}



extern "C"
__global__ void bm_phase_exp_times_scalar_range(double *a, double *b, double c, int *d , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = exp(c * b[i]) * d[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = exp(c * b[i]) * d[i];
        }
    }
    ;
}



extern "C"
__global__ void bm_phase_mul_add_range(double *a, double b, double *c, double d , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = b * c[i] + d;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = b * c[i] + d;
        }
    }
    ;
}



extern "C"
__global__ void bm_sin_cos_range(double *a, double *b, double *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            sincos(a[i], &b[i], &c[i]);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            sincos(a[i], &b[i], &c[i]);
        }
    }
    ;
}



extern "C"
__global__ void d_multiply_range(double *a, double *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] *= b[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] *= b[i];
        }
    }
    ;
}



extern "C"
__global__ void d_multscalar_range(double *a, double *b, double c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = c * b[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = c * b[i];
        }
    }
    ;
}



extern "C"
__global__ void scale_kernel_int_range(int a, int *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    ;
}



extern "C"
__global__ void scale_kernel_double_range(double a, double *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    ;
}

extern "C"
__global__ void synchrotron_radiation(
    double *  beam_dE,
    const double U0,
    const int n_macroparticles,
    const double tau_z,
    const int n_kicks)
{

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const double const_synch_rad = 2.0 / tau_z;

    for (int j = 0; j < n_kicks; j++) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x)
            beam_dE[i] -= const_synch_rad * beam_dE[i] + U0;
    }
}

extern "C"
__global__ void synchrotron_radiation_full(
    double *  beam_dE,
    const double U0,
    const int n_macroparticles,
    const double sigma_dE,
    const double tau_z,
    const double energy,
    const int n_kicks
)
{   unsigned int seed = 0;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const double const_quantum_exc = 2.0 * sigma_dE / sqrt(tau_z) * energy;
    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);
    const double const_synch_rad = 2.0 / tau_z;
    for (int j = 0; j < n_kicks; j++) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x)
            beam_dE[i] -= const_synch_rad * beam_dE[i] + U0 - const_quantum_exc * hiprand_normal_double(&state);
    }
}
