#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <hiprand/hiprand_kernel.h>
#define REDUCE(a, b) (a+b)
#define BLOCK_SIZE 512
#define PI 3.141592653589793238462643383279502884197169399375105820974944592307816406286
#define PI_DIV_2 3.141592653589793238462643383279502884197169399375105820974944592307816406286/2

// Note that any atomic operation can be implemented based on atomicCAS() (Compare And Swap). 
// For example, atomicAdd() for double-precision floating-point numbers is not 
// available on devices with compute capability lower than 6.0 but it can be implemented 
// as follows: 
// #if __CUDA_ARCH__ < 600
// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                               (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;

//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));

//     // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//     } while (assumed != old);

//     return __longlong_as_double(old);
// }
// #endif


extern "C"
__global__ void gpu_losses_longitudinal_cut(
    float *dt,
    float *dev_id,
    const int size,
    const float min_dt,
    const float max_dt)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if ((dt[i] - min_dt) * (max_dt - dt[i]) < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void gpu_losses_energy_cut(
    float *dE,
    float *dev_id,
    const int size,
    const float min_dE,
    const float max_dE)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if ((dE[i] - min_dE) * (max_dE - dE[i]) < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void gpu_losses_below_energy(
    float *dE,
    float *dev_id,
    const int size,
    const float min_dE)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
        if (dE[i] - min_dE < 0)
            dev_id[i] = 0;
}

extern "C"
__global__ void cuinterp(float *x,
                         int x_size,
                         float *xp,
                         int xp_size,
                         float *yp,
                         float *y,
                         float left,
                         float right)
{
    if (left == 0.12345)
        left = yp[0];
    if (right == 0.12345)
        right = yp[xp_size - 1];
    float curr;
    int lo;
    int mid;
    int hi;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < x_size; i += blockDim.x * gridDim.x) {
        //need to find the right bin with binary search
        // looks like bisect_left
        curr = x[i];
        hi = xp_size;
        lo = 0;
        while (lo < hi) {
            mid = (lo + hi) / 2;
            if (xp[mid] < curr)
                lo = mid + 1;
            else
                hi = mid;
        }
        if (lo == xp_size)
            y[i] = right;
        else if (xp[lo - 1] == curr)
            y[i] = yp[i];
        else if (lo <= 1)
            y[i] = left;
        else {
            y[i] = yp[lo - 1] +
                   (yp[lo] - yp[lo - 1]) * (x[i] - xp[lo - 1]) /
                   (xp[lo] - xp[lo - 1]);
        }

    }
}

extern "C"
__global__ void cugradient(
    float x,
    int *y,
    float *g,
    int size)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid + 1; i < size - 1; i += blockDim.x * gridDim.x) {

        g[i] = (y[i + 1] - y[i - 1]) / (2 * x);
        // g[i] = (hs*hs*fd + (hd*hd-hs*hs)*fx - hd*hd*fs)/
        //     (hs*hd*(hd+hs));
    }
    if (tid == 0)
        g[0] = (y[1] - y[0]) / x;
    if (tid == 32)
        g[size - 1] = (y[size - 1] - y[size - 2]) / x;
}


extern "C"
__global__ void gpu_beam_fb_track_other(float *omega_rf,
                                        float *harmonic,
                                        float *dphi_rf,
                                        float *omega_rf_d,
                                        float *phi_rf,
                                        // float pi,
                                        float domega_rf,
                                        int size,
                                        int counter,
                                        int n_rf)
{
    float a, b, c;
    for (int i = threadIdx.x; i < n_rf; i += blockDim.x) {
        a = domega_rf * harmonic[i * size + counter] / harmonic[counter];
        b =  2.0 * PI * harmonic[size * i + counter] * (a + omega_rf[i * size + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        c = dphi_rf[i] +  b;
        omega_rf[i * size + counter] += a;
        dphi_rf[i] +=  b;
        phi_rf[size * i + counter] += c;
    }
}

extern "C"
__global__ void gpu_rf_voltage_calc_mem_ops(float *new_voltages,
        float *new_omega_rf,
        float *new_phi_rf,
        float *voltages,
        float *omega_rf,
        float *phi_rf,
        int start,
        int end,
        int step)
{
    int idx = 0;
    for (int i = threadIdx.x * step + start; i < end; i += blockDim.x * step) {
        new_voltages[idx] = voltages[i];
        new_omega_rf[idx] = omega_rf[i];
        new_phi_rf[idx] = phi_rf[i];
        idx++;
    }
} 

extern "C"
__global__ void halve_edges(float *my_array, int size) {
    //__shared__ my_sum;
    int tid = threadIdx.x;
    if (tid == 0) {
        my_array[0] = my_array[0] / 2.;
    }
    if (tid == 32) {
        my_array[size - 1] = my_array[size - 1] / 2.;
    }
}

extern "C"
__global__ void simple_kick(
    const float  *beam_dt,
    float        *beam_dE,
    const int n_rf,
    const float  *voltage,
    const float  *omega_RF,
    const float  *phi_RF,
    const int n_macroparticles,
    const float acc_kick
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float my_beam_dt;
    float sin_res;
    float dummy;
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        my_beam_dt = beam_dt[i];
        for (int j = 0; j < n_rf; j++) {
            sincosf(omega_RF[j]*my_beam_dt + phi_RF[j], &sin_res, &dummy);
            beam_dE[i] += voltage[j] * sin_res;
        }
        beam_dE[i] += acc_kick;
    }
}

extern "C"
__global__ void rf_volt_comp(float *voltage,
                             float *omega_rf,
                             float *phi_rf,
                             float *bin_centers,
                             int n_rf,
                             int n_bins,
                             int f_rf,
                             float *rf_voltage)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__ float s[];
    if (threadIdx.x == 0){
        for (int j = 0; j < n_rf; j++) {
            s[j] = voltage[j];
            s[j + n_rf] = omega_rf[j];
            s[j + 2 * n_rf] = phi_rf[j];
        }
    }

    __syncthreads();
    for (int i = tid; i < n_bins; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_rf; j++)
            // rf_voltage[i] = voltage[j] * sin(omega_rf[j] * bin_centers[i] + phi_rf[j]);
            rf_voltage[i] = s[j] * sinf(s[j+n_rf] * bin_centers[i] + s[j+2*n_rf]);
    }
}


extern "C"
__global__ void drift(float *beam_dt,
        const float  *beam_dE,
        const int solver,
        const float T0, const float length_ratio,
        const float alpha_order, const float eta_zero,
        const float eta_one, const float eta_two,
        const float alpha_zero, const float alpha_one,
        const float alpha_two,
        const float beta, const float energy,
        const int n_macroparticles)
{
    float T = T0 * length_ratio;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if ( solver == 0 )
    {
        float coeff = eta_zero / (beta * beta * energy);
        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
            beam_dt[i] += T * coeff * beam_dE[i];
    }

    else if ( solver == 1 )
    {
        const float coeff = 1. / (beta * beta * energy);
        const float eta0 = eta_zero * coeff;
        const float eta1 = eta_one * coeff * coeff;
        const float eta2 = eta_two * coeff * coeff * coeff;

        if (alpha_order == 0)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) - 1.);
        else if (alpha_order == 1)
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]) - 1.);
        else
            for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
                beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]
                                         - eta1 * beam_dE[i] * beam_dE[i]
                                         - eta2 * beam_dE[i] * beam_dE[i] * beam_dE[i]) - 1.);
    }

    else
    {

        const float invbetasq = 1 / (beta * beta);
        const float invenesq = 1 / (energy * energy);
        // float beam_delta;

        
        for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)

        {

            float beam_delta = sqrt(1. + invbetasq *
                              (beam_dE[i] * beam_dE[i] * invenesq + 2.*beam_dE[i] / energy)) - 1.;

            beam_dt[i] += T * (
                              (1. + alpha_zero * beam_delta +
                               alpha_one * (beam_delta * beam_delta) +
                               alpha_two * (beam_delta * beam_delta * beam_delta)) *
                              (1. + beam_dE[i] / energy) / (1. + beam_delta) - 1.);

        }

    }    
    
}   




extern "C"
__global__ void histogram(float * input,
                          int * output, const float cut_left,
                          const float cut_right, const int n_slices,
                          const int n_macroparticles)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    float const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i = i + blockDim.x * gridDim.x) {
        target_bin = floorf((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        atomicAdd(&(output[target_bin]), 1);
    }
}

extern "C"
__global__ void hybrid_histogram(float * input,
                                 int * output, const float cut_left,
                                 const float cut_right, const unsigned int n_slices,
                                 const int n_macroparticles, const int capacity)
{
    extern __shared__ int block_hist[];
    //reset shared memory
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    float const inv_bin_width = n_slices / (cut_right - cut_left);

    const int low_tbin = (n_slices / 2) - (capacity / 2);
    const int high_tbin = low_tbin + capacity;


    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floorf((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        if (target_bin >= low_tbin && target_bin < high_tbin)
            atomicAdd(&(block_hist[target_bin - low_tbin]), 1);
        else
            atomicAdd(&(output[target_bin]), 1);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        atomicAdd(&output[low_tbin + i], block_hist[i]);
}


extern "C"
__global__ void sm_histogram(float * input,
                             int * output, const float cut_left,
                             const float cut_right, const unsigned int n_slices,
                             const int n_macroparticles)
{
    extern __shared__ int block_hist[];
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    float const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floorf((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        atomicAdd(&(block_hist[target_bin]), 1);
    }
    __syncthreads();
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        atomicAdd(&output[i], block_hist[i]);
}


extern "C"
__global__ void lik_only_gm_copy(
    float *beam_dt,
    float *beam_dE,
    const float *voltage_array,
    const float *bin_centers,
    const float charge,
    const int n_slices,
    const int n_macroparticles,
    const float acc_kick,
    float *glob_voltageKick,
    float *glob_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);


    for (int i = tid; i < n_slices - 1; i += gridDim.x * blockDim.x) {
        glob_voltageKick[i] = charge * (voltage_array[i + 1] - voltage_array[i])
                              * inv_bin_width;
        glob_factor[i] = (charge * voltage_array[i] - bin_centers[i] * glob_voltageKick[i])
                         + acc_kick;
    }
}


extern "C"
__global__ void lik_only_gm_comp(
    float *beam_dt,
    float *beam_dE,
    const float *voltage_array,
    const float *bin_centers,
    const float charge,
    const int n_slices,
    const int n_macroparticles,
    const float acc_kick,
    float *glob_voltageKick,
    float *glob_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    int fbin;
    const float bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = floorf((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1) && (fbin >= 0))
            beam_dE[i] += beam_dt[i] * glob_voltageKick[fbin] + glob_factor[fbin];
    }
}


extern "C"
__global__ void lik_drift_only_gm_comp(
    float *beam_dt,
    float *beam_dE,
    const float *voltage_array,
    const float *bin_centers,
    const float charge,
    const int n_slices,
    const int n_macroparticles,
    const float acc_kick,
    float *glob_voltageKick,
    float *glob_factor,
    const float T0, const float length_ratio,
    const float eta0, const float beta, const float energy
)
{
    const float T = T0 * length_ratio * eta0 / (beta * beta * energy);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    unsigned fbin;
    const float bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = (unsigned) floorf((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1))
            beam_dE[i] += beam_dt[i] * glob_voltageKick[fbin] + glob_factor[fbin];
        // beam_dt[i] += T * (1. / (1. - eta0 * beam_dE[i]) -1.);
        beam_dt[i] += T * beam_dE[i];
    }
}

extern "C"
__global__ void beam_phase_v2(
    const float *bin_centers,
    const int *profile,
    const float alpha,
    const float *omega_rf_ar,
    const float *phi_rf_ar,
    const int ind,
    const float bin_size,
    float *array1,
    float *array2,
    const int n_bins)
{
    float omega_rf = omega_rf_ar[ind];
    float phi_rf = phi_rf_ar[ind];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float a, b;
    float sin_res, cos_res;
    for (int i = tid; i < n_bins; i += gridDim.x * blockDim.x) {
        a = omega_rf * bin_centers[i] + phi_rf;
        sincosf(a, &sin_res, &cos_res);
        b = expf(alpha * bin_centers[i]) * profile[i];
        array1[i] = b * sin_res;
        array2[i] = b * cos_res;
        // array1[i] = a;
        // array2[i] = b;

    }
} 

extern "C" 
__global__ void beam_phase_sum(
    const float *ar1,
    const float *ar2,
    float *scoeff,
    float *coeff,
    int n_bins)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid == 0) {
        scoeff[0] = 0;
        coeff[0] = 0;
    }
    float my_sum_1 = 0;
    float my_sum_2 = 0;
    if (tid == 0) {
        my_sum_1 += ar1[0] / 2 + ar1[n_bins - 1] / 2;
        my_sum_2 += ar2[0] / 2 + ar2[n_bins - 1] / 2;
    }
    for (int i = tid + 1; i < n_bins - 1; i += gridDim.x * blockDim.x) {
        my_sum_1 += ar1[i];
        my_sum_2 += ar2[i];
    }
    atomicAdd(&(scoeff[0]), my_sum_1);
    atomicAdd(&(coeff[0]), my_sum_2);
    __syncthreads();
    if (tid == 0)
        scoeff[0] = scoeff[0] / coeff[0];

} 

extern "C"
__global__ void gpu_trapz_custom(
    float *y,
    float x,
    int sz,
    float *res)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    float my_sum = 0;
    for (int i = tid; i < sz - 1; i += gridDim.x * blockDim.x)
        my_sum += (y[i] + y[i + 1]) * x / 2.0;

    atomicAdd(&(res[0]), my_sum);
}


extern "C" 
__global__ void gpu_trapz_stage1(float *out, float *y, float x, int sz,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + ((i < sz - 1) ? x * (y[i] + y[i + 1]) / 2.0 : 0.0);
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}



extern "C" 
__global__ void gpu_trapz_stage2(float *out, const float *pycuda_reduction_inp, float *y, float x, int sz,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + (pycuda_reduction_inp[i]);
        i = 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}


extern "C"
__global__ void mean_non_zeros_stage1(float *out, float *x, float *id,
                           unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = acc + ((id[i] != 0) * x[i]);
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = sdata[tid] + sdata[tid + 256]; }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = sdata[tid] + sdata[tid + 128]; }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = sdata[tid] + sdata[tid + 64]; }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (512 >= 64) smem[tid] = smem[tid] + smem[tid + 32];
        if (512 >= 32) smem[tid] = smem[tid] + smem[tid + 16];
        if (512 >= 16) smem[tid] = smem[tid] + smem[tid + 8];
        if (512 >= 8)  smem[tid] = smem[tid] + smem[tid + 4];
        if (512 >= 4)  smem[tid] = smem[tid] + smem[tid + 2];
        if (512 >= 2)  smem[tid] = smem[tid] + smem[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void mean_non_zeros_stage2(float *out, const float *pycuda_reduction_inp, float *x, float *id,
                           unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 512 * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += 512;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (512 >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (512 >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (512 >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (512 >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (512 >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (512 >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (512 >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (512 >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (512 >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void stdKernel_stage1(float *out, float *x, float *y, float m,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, ((y[i] != 0) * (x[i] - m) * (x[i] - m)));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void stdKernel_stage2(float *out, const float *pycuda_reduction_inp, float *x, float *y, float m,
                      unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void sum_non_zeros_stage1(float *out, float *x,
                          unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, ((x[i] != 0)));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}




extern "C"
__global__ void sum_non_zeros_stage2(float *out, const float *pycuda_reduction_inp, float *x,
                          unsigned int seq_count, unsigned int n)
{
    // Needs to be variable-size to prevent the braindead CUDA compiler from
    // running constructors on this array. Grrrr.
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * BLOCK_SIZE * seq_count + tid;
    float acc = 0;
    for (unsigned s = 0; s < seq_count; ++s)
    {
        if (i >= n)
            break;
        acc = REDUCE(acc, (pycuda_reduction_inp[i]));
        i += BLOCK_SIZE;
    }
    sdata[tid] = acc;
    __syncthreads();
#if (BLOCK_SIZE >= 512)
    if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 256)
    if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
    __syncthreads();
#endif
#if (BLOCK_SIZE >= 128)
    if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
    __syncthreads();
#endif
    if (tid < 32)
    {
        // 'volatile' required according to Fermi compatibility guide 1.2.2
        volatile float *smem = sdata;
        if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
        if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
        if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
        if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
        if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
        if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}



extern "C"
__global__ void gpu_copy_i2d(float *x, int *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = (float) y[i] * 1.0;
    }
    ;
}



extern "C"
__global__ void gpu_copy_d2d(float *x, float *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i];
    }
    ;
}



extern "C"
__global__ void gpu_complex_copy(pycuda::complex<float> *x, pycuda::complex<float> *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i];
    }
    ;
}



extern "C"
__global__ void gpu_diff(int *a, float *b, float c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] = (a[i + 1] - a[i]) / c;
    }
    ;
}


extern "C"
__global__ void set_zero_float(float *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}

extern "C"
__global__ void set_zero_double(double *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}



extern "C"
__global__ void set_zero_int(int *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}



extern "C"
__global__ void set_zero_complex64(pycuda::complex<float> *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}


extern "C"
__global__ void set_zero_complex128(pycuda::complex<double> *x, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = 0;
    }
    ;
}


extern "C"
__global__ void increase_by_value(float *x, float a, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] += a;
    }
    ;
}



extern "C"
__global__ void add_array(float *x, float *y, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] += y[i];
    }
    ;
}



extern "C"
__global__ void complex_mul(pycuda::complex<float> *x, pycuda::complex<float> *y, pycuda::complex<float> *z, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        z[i] = x[i] * y[i];
    }
    ;
}



extern "C"
__global__ void gpu_mul(float *x, float *y, float a, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = a * y[i];
    }
    ;
}



extern "C"
__global__ void gpu_copy_one(float *x, float *y, int ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[ind];
    }
    ;
}



extern "C"
__global__ void first_kernel_x(float *omega_rf, float *harmonic,  float domega_rf, int size, int counter, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
    }
    ;
}



extern "C"
__global__ void second_kernel_x(float *dphi_rf, float *harmonic, float *omega_rf, float *omega_rf_d, int size, int counter, float pi, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
    }
    ;
}



extern "C"
__global__ void third_kernel_x(float *x, float *y, int size_0, int counter, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i * size_0 + counter] += y[i];
    }
    ;
}



extern "C"
__global__ void indexing_double(float *out, float *in, int *ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        out[i] = in[ind[i]];
    }
    ;
}


// extern "C"
// __global__ void indexing_float(float *out, float *in, int *ind, long n)
// {
//     unsigned tid = threadIdx.x;
//     unsigned total_threads = gridDim.x * blockDim.x;
//     unsigned cta_start = blockDim.x * blockIdx.x;
//     unsigned i;
//     ;
//     for (i = cta_start + tid; i < n; i += total_threads)
//     {
//         out[i] = in[ind[i]];
//     }
//     ;
// }



extern "C"
__global__ void indexing_int(float *out, int *in, int *ind, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        out[i] = in[ind[i]];
    }
    ;
}



extern "C"
__global__ void sincos_mul_add(float *ar, float a, float b, float *s, float *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        sincosf(a * ar[i] + b, &s[i], &c[i]);
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_2(float *ar, float a, float b, float *s, float *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        s[i] = cosf(a * ar[i] + b - PI_DIV_2); c[i] = cosf(a * ar[i] + b);
    }
    ;
}



extern "C"
__global__ void mul_d(float *a1, float *a2, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a1[i] *= a2[i];
    }
    ;
}



extern "C"
__global__ void add_kernel(float *a, float *b, float *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = b[i] + c[i];
    }
    ;
}



extern "C"
__global__ void first_kernel_tracker(float *phi_rf, float x, float *phi_noise, int len, int turn, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
    }
    ;
}



extern "C"
__global__ void second_kernel_tracker(float *phi_rf, float *omega_rf, float *phi_mod0, float *phi_mod1, int size, int turn, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
    }
    ;
}



extern "C"
__global__ void copy_column(float *x, float *y, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i * size + column];
    }
    ;
}



extern "C"
__global__ void rf_voltage_calculation_kernel(float *x, float *y, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        x[i] = y[i * size + column];
    }
    ;
}



extern "C"
__global__ void cavityFB_case(float *rf_voltage, float *voltage, float *omega_rf, float *phi_rf, float *bin_centers, float V_corr, float phi_corr, int size, int column, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        rf_voltage[i] = voltage[0] * V_corr * sinf(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
    }
    ;
}



extern "C"
__global__ void bm_phase_exp_times_scalar(float *a, float *b, float c, int *d, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = expf(c * b[i]) * d[i];
    }
    ;
}



extern "C"
__global__ void bm_phase_mul_add(float *a, float b, float *c, float d, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = b * c[i] + d;
    }
    ;
}



extern "C"
__global__ void bm_sin_cos(float *a, float *b, float *c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        sincosf(a[i], &b[i], &c[i]);
    }
    ;
}



extern "C"
__global__ void d_multiply(float *a, float *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] *= b[i];
    }
    ;
}



extern "C"
__global__ void d_multscalar(float *a, float *b, float c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] = c * b[i];
    }
    ;
}


extern "C"
__global__ void d_mul_int_by_scalar(int *a, float c, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        a[i] *= c;
    }
    ;
}


extern "C"
__global__ void scale_kernel_int(int a, int *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}



extern "C"
__global__ void scale_kernel_double(double a, double *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}


extern "C"
__global__ void scale_kernel_float(float a, float *b, long n)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    unsigned i;
    ;
    for (i = cta_start + tid; i < n; i += total_threads)
    {
        b[i] /= a ;
    }
    ;
}


extern "C"
__global__ void gpu_copy_i2d_range(float *x, int *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = (float) y[i] * 1.0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = (float) y[i] * 1.0;
        }
    }
    ;
}



extern "C"
__global__ void gpu_copy_d2d_range(float *x, float *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_complex_copy_range(pycuda::complex<float> *x, pycuda::complex<float> *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_diff_range(int *a, float *b, float c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] = (a[i + 1] - a[i]) / c;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] = (a[i + 1] - a[i]) / c;
        }
    }
    ;
}


extern "C"
__global__ void set_zero_double_range(double *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}

extern "C"
__global__ void set_zero_float_range(float *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}



extern "C"
__global__ void set_zero_int_range(int *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}


extern "C"
__global__ void set_zero_complex64_range(pycuda::complex<float> *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}

extern "C"
__global__ void set_zero_complex128_range(pycuda::complex<double> *x , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = 0;
        }
    }
    ;
}



extern "C"
__global__ void increase_by_value_range(float *x, float a , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] += a;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] += a;
        }
    }
    ;
}



extern "C"
__global__ void add_array_range(float *x, float *y , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] += y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] += y[i];
        }
    }
    ;
}



extern "C"
__global__ void complex_mul_range(pycuda::complex<float> *x, pycuda::complex<float> *y, pycuda::complex<float> *z , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            z[i] = x[i] * y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            z[i] = x[i] * y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_mul_range(float *x, float *y, float a , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = a * y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = a * y[i];
        }
    }
    ;
}



extern "C"
__global__ void gpu_copy_one_range(float *x, float *y, int ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[ind];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[ind];
        }
    }
    ;
}



extern "C"
__global__ void first_kernel_x_range(float *omega_rf, float *harmonic,  float domega_rf, int size, int counter , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            omega_rf[i * size + counter] += domega_rf * harmonic[i * size + counter] / harmonic[counter];
        }
    }
    ;
}



extern "C"
__global__ void second_kernel_x_range(float *dphi_rf, float *harmonic, float *omega_rf, float *omega_rf_d, int size, int counter, float pi , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            dphi_rf[i] +=  2.0 * pi * harmonic[size * i + counter] * (omega_rf[size * i + counter] - omega_rf_d[size * i + counter]) / omega_rf_d[size * i + counter];
        }
    }
    ;
}



extern "C"
__global__ void third_kernel_x_range(float *x, float *y, int size_0, int counter , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i * size_0 + counter] += y[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i * size_0 + counter] += y[i];
        }
    }
    ;
}


extern "C"
__global__ void indexing_double_range(double *out, double *in, int *ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    ;
}


extern "C"
__global__ void indexing_float_range(float *out, float *in, int *ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    ;
}



extern "C"
__global__ void indexing_int_range(float *out, int *in, int *ind , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            out[i] = in[ind[i]];
        }
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_range(float *ar, float a, float b, float *s, float *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            sincosf(a * ar[i] + b, &s[i], &c[i]);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            sincosf(a * ar[i] + b, &s[i], &c[i]);
        }
    }
    ;
}



extern "C"
__global__ void sincos_mul_add_2_range(float *ar, float a, float b, float *s, float *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            s[i] = cosf(a * ar[i] + b - PI_DIV_2); c[i] = cosf(a * ar[i] + b);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            s[i] = cosf(a * ar[i] + b - PI_DIV_2); c[i] = cosf(a * ar[i] + b);
        }
    }
    ;
}



extern "C"
__global__ void mul_d_range(float *a1, float *a2 , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a1[i] *= a2[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a1[i] *= a2[i];
        }
    }
    ;
}



extern "C"
__global__ void add_kernel_range(float *a, float *b, float *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = b[i] + c[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = b[i] + c[i];
        }
    }
    ;
}



extern "C"
__global__ void first_kernel_tracker_range(float *phi_rf, float x, float *phi_noise, int len, int turn , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            phi_rf[len * i + turn] += x * phi_noise[len * i + turn];
        }
    }
    ;
}



extern "C"
__global__ void second_kernel_tracker_range(float *phi_rf, float *omega_rf, float *phi_mod0, float *phi_mod1, int size, int turn , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            phi_rf[i * size + turn] += phi_mod0[i * size + turn]; omega_rf[i * size + turn] += phi_mod1[i * size + turn];
        }
    }
    ;
}



extern "C"
__global__ void copy_column_range(float *x, float *y, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    ;
}



extern "C"
__global__ void rf_voltage_calculation_kernel_range(float *x, float *y, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            x[i] = y[i * size + column];
        }
    }
    ;
}



extern "C"
__global__ void cavityFB_case_range(float *rf_voltage, float *voltage, float *omega_rf, float *phi_rf, float *bin_centers, float V_corr, float phi_corr, int size, int column , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            rf_voltage[i] = voltage[0] * V_corr * sinf(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            rf_voltage[i] = voltage[0] * V_corr * sinf(omega_rf[0] * bin_centers[i] + phi_rf[0] + phi_corr);
        }
    }
    ;
}



extern "C"
__global__ void bm_phase_exp_times_scalar_range(float *a, float *b, float c, int *d , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = expf(c * b[i]) * d[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = expf(c * b[i]) * d[i];
        }
    }
    ;
}



extern "C"
__global__ void bm_phase_mul_add_range(float *a, float b, float *c, float d , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = b * c[i] + d;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = b * c[i] + d;
        }
    }
    ;
}



extern "C"
__global__ void bm_sin_cos_range(float *a, float *b, float *c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            sincosf(a[i], &b[i], &c[i]);
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            sincosf(a[i], &b[i], &c[i]);
        }
    }
    ;
}



extern "C"
__global__ void d_multiply_range(float *a, float *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] *= b[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] *= b[i];
        }
    }
    ;
}



extern "C"
__global__ void d_multscalar_range(float *a, float *b, float c , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            a[i] = c * b[i];
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            a[i] = c * b[i];
        }
    }
    ;
}



extern "C"
__global__ void scale_kernel_int_range(int a, int *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    ;
}



extern "C"
__global__ void scale_kernel_float_range(float a, float *b , long start, long stop, long step)
{
    unsigned tid = threadIdx.x;
    unsigned total_threads = gridDim.x * blockDim.x;
    unsigned cta_start = blockDim.x * blockIdx.x;
    long i;
    ;
    if (step < 0)
    {
        for (i = start + (cta_start + tid) * step;
                i > stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    else
    {
        for (i = start + (cta_start + tid) * step;
                i < stop; i += total_threads * step)
        {
            b[i] /= a ;
        }
    }
    ;
}

extern "C"
__global__ void synchrotron_radiation(
    float *  beam_dE,
    const float U0,
    const int n_macroparticles,
    const float tau_z,
    const int n_kicks)
{

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const float const_synch_rad = 2.0 / tau_z;

    for (int j = 0; j < n_kicks; j++) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x)
            beam_dE[i] -= const_synch_rad * beam_dE[i] + U0;
    }
}

extern "C"
__global__ void synchrotron_radiation_full(
    float *  beam_dE,
    const float U0,
    const int n_macroparticles,
    const float sigma_dE,
    const float tau_z,
    const float energy,
    const int n_kicks
)
{   unsigned int seed = 0;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const float const_quantum_exc = 2.0 * sigma_dE / sqrtf(tau_z) * energy;
    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);
    const float const_synch_rad = 2.0 / tau_z;
    for (int j = 0; j < n_kicks; j++) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x)
            beam_dE[i] -= const_synch_rad * beam_dE[i] + U0 - const_quantum_exc * hiprand_normal(&state);
    }
}
